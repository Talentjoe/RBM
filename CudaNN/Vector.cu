#include "hip/hip_runtime.h"
#include "Vector.cuh"

#include <iostream>
#include "CudaFunctions.cuh"

namespace NN {
    void Vector::initRandom(float max, float min) {
        GetRand(*this,max,min);
    }

    void Vector::resize(int s) {
        if (elements != nullptr) {
            delete[] elements;
        }
        size = s;
        elements = new float[s];
        hipMalloc(&d_elements, sizeof(float) * s);
    }

    void Vector::cpDtoH() const {
        hipMemcpy(elements, d_elements, sizeof(float) * size, hipMemcpyDeviceToHost);
    }

    void Vector::cpDtoHAsync() const {
        hipMemcpyAsync(elements, d_elements, sizeof(float) * size, hipMemcpyDeviceToHost);
    }

    void Vector::cpHoD() const {
        hipMemcpy(d_elements, elements, sizeof(float) * size, hipMemcpyHostToDevice);
    }

    void Vector::cpHoDAsync() const {
        hipMemcpyAsync(d_elements, elements, sizeof(float) * size, hipMemcpyHostToDevice);
    }

    void Vector::free() const {
        hipFree(d_elements);
    }

    void Vector::printVec() const {
        for (int i = 0; i < size; i++) {
            std::cout << elements[i] << " ";
        }
        std::cout << std::endl;
    }
}
