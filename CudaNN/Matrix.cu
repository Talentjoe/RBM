#include "hip/hip_runtime.h"
#include "Matrix.cuh"

#include <iostream>
#include "CudaFunctions.cuh"

namespace NN {
    void Matrix::initRand(float max, float min) const {
        GetRand(*this,max,min);
    }

    void Matrix::resize(int w, int h) {
        width = w;
        height = h;
        elements = new float[w * h];
        hipMalloc(&d_elements, sizeof(float) * w * h);
    }

    void Matrix::cpDtoH() const {
        hipMemcpy(elements, d_elements, sizeof(float) * width * height, hipMemcpyDeviceToHost);
    }

    void Matrix::cpDtoHAsync() const {
        hipMemcpyAsync(elements, d_elements, sizeof(float) * width * height, hipMemcpyDeviceToHost);
    }

    void Matrix::cpHoD() const {
        hipMemcpy(d_elements, elements, sizeof(float) * width * height, hipMemcpyHostToDevice);
    }

    void Matrix::cpHoDAsync() const {
        hipMemcpyAsync(d_elements, elements, sizeof(float) * width * height, hipMemcpyHostToDevice);
    }

    void Matrix::free() const {
        delete[] elements;
        hipFree(d_elements);
    }

    void Matrix::printMat() const {
            for (int j = 0; j < width; j++) {
        for (int i = 0; i < height; i++) {
                std::cout << elements[i * width + j] << " ";
            }
            std::cout << std::endl;
        }
    }
}
